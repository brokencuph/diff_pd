#include "hip/hip_runtime.h"
#include "deformable_cuda.h"

#include <cstdlib>
#include <algorithm>
#include <numeric>
#include <iterator>
#include <iostream>

using namespace std;

__global__ void addWithCuda(const int* a, const int* b, int* c)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = max(a[idx], b[idx]);
}

void test_add()
{
    const int N = 100;
    int* host_a = new int[N];
    int* host_b = new int[N];
    int* host_c = new int[N];

    iota(host_a, host_a + N, 0);
    iota(host_b, host_b + N, 10);

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);

    addWithCuda<<<N, 1>>>(dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    copy(host_c, host_c + N, ostream_iterator<int>(cout, " "));
}